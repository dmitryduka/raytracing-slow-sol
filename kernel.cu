#include "hip/hip_runtime.h"
﻿#include <spdlog/spdlog.h>
#include <SDL.h>
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <vector>

namespace
{
    constexpr auto WINDOW_WIDTH = 1024;
    constexpr auto WINDOW_HEIGHT = 1024;
    constexpr auto FB_WIDTH = WINDOW_WIDTH;
    constexpr auto FB_HEIGHT = WINDOW_HEIGHT;
    constexpr auto HORIZONTAL_FOV = 90.0f;
    const dim3 THREADS{ 16, 16 };
    const dim3 BLOCKS{ FB_WIDTH / THREADS.x, FB_HEIGHT / THREADS.y };
}

namespace math
{
    struct mat4 {
        float m[4][4];

        __device__ mat4() {
            identity();
        }

        __device__ mat4(
            const float m11, const float m12, const float m13, const float m14,
            const float m21, const float m22, const float m23, const float m24,
            const float m31, const float m32, const float m33, const float m34,
            const float m41, const float m42, const float m43, const float m44
        ) {
            m[0][0] = m11; m[1][0] = m12; m[2][0] = m13; m[3][0] = m14;
            m[0][1] = m21; m[1][1] = m22; m[2][1] = m23; m[3][1] = m24;
            m[0][2] = m31; m[1][2] = m32; m[2][2] = m33; m[3][2] = m34;
            m[0][3] = m41; m[1][3] = m42; m[2][3] = m43; m[3][3] = m44;
        }

        __device__ float* operator[] (const size_t idx) {
            return m[idx];
        }

        __device__ float4 operator*(const float4& v) const {
            float4 ret;
            ret.x = m[0][0] * v.x + m[1][0] * v.y + m[2][0] * v.z + m[3][0] * v.w;
            ret.y = m[0][1] * v.x + m[1][1] * v.y + m[2][1] * v.z + m[3][1] * v.w;
            ret.z = m[0][2] * v.x + m[1][2] * v.y + m[2][2] * v.z + m[3][2] * v.w;
            ret.w = m[0][3] * v.x + m[1][3] * v.y + m[2][3] * v.z + m[3][3] * v.w;
            return ret;
        }

        __device__ mat4 inverse() const {
            const auto n11 = m[0][0], n12 = m[1][0], n13 = m[2][0], n14 = m[3][0];
            const auto n21 = m[0][1], n22 = m[1][1], n23 = m[2][1], n24 = m[3][1];
            const auto n31 = m[0][2], n32 = m[1][2], n33 = m[2][2], n34 = m[3][2];
            const auto n41 = m[0][3], n42 = m[1][3], n43 = m[2][3], n44 = m[3][3];

            const auto t11 = n23 * n34 * n42 - n24 * n33 * n42 + n24 * n32 * n43 - n22 * n34 * n43 - n23 * n32 * n44 + n22 * n33 * n44;
            const auto t12 = n14 * n33 * n42 - n13 * n34 * n42 - n14 * n32 * n43 + n12 * n34 * n43 + n13 * n32 * n44 - n12 * n33 * n44;
            const auto t13 = n13 * n24 * n42 - n14 * n23 * n42 + n14 * n22 * n43 - n12 * n24 * n43 - n13 * n22 * n44 + n12 * n23 * n44;
            const auto t14 = n14 * n23 * n32 - n13 * n24 * n32 - n14 * n22 * n33 + n12 * n24 * n33 + n13 * n22 * n34 - n12 * n23 * n34;

            const auto det = n11 * t11 + n21 * t12 + n31 * t13 + n41 * t14;
            const auto idet = 1.0f / det;

            mat4 ret;

            ret[0][0] = t11 * idet;
            ret[0][1] = (n24 * n33 * n41 - n23 * n34 * n41 - n24 * n31 * n43 + n21 * n34 * n43 + n23 * n31 * n44 - n21 * n33 * n44) * idet;
            ret[0][2] = (n22 * n34 * n41 - n24 * n32 * n41 + n24 * n31 * n42 - n21 * n34 * n42 - n22 * n31 * n44 + n21 * n32 * n44) * idet;
            ret[0][3] = (n23 * n32 * n41 - n22 * n33 * n41 - n23 * n31 * n42 + n21 * n33 * n42 + n22 * n31 * n43 - n21 * n32 * n43) * idet;

            ret[1][0] = t12 * idet;
            ret[1][1] = (n13 * n34 * n41 - n14 * n33 * n41 + n14 * n31 * n43 - n11 * n34 * n43 - n13 * n31 * n44 + n11 * n33 * n44) * idet;
            ret[1][2] = (n14 * n32 * n41 - n12 * n34 * n41 - n14 * n31 * n42 + n11 * n34 * n42 + n12 * n31 * n44 - n11 * n32 * n44) * idet;
            ret[1][3] = (n12 * n33 * n41 - n13 * n32 * n41 + n13 * n31 * n42 - n11 * n33 * n42 - n12 * n31 * n43 + n11 * n32 * n43) * idet;

            ret[2][0] = t13 * idet;
            ret[2][1] = (n14 * n23 * n41 - n13 * n24 * n41 - n14 * n21 * n43 + n11 * n24 * n43 + n13 * n21 * n44 - n11 * n23 * n44) * idet;
            ret[2][2] = (n12 * n24 * n41 - n14 * n22 * n41 + n14 * n21 * n42 - n11 * n24 * n42 - n12 * n21 * n44 + n11 * n22 * n44) * idet;
            ret[2][3] = (n13 * n22 * n41 - n12 * n23 * n41 - n13 * n21 * n42 + n11 * n23 * n42 + n12 * n21 * n43 - n11 * n22 * n43) * idet;

            ret[3][0] = t14 * idet;
            ret[3][1] = (n13 * n24 * n31 - n14 * n23 * n31 + n14 * n21 * n33 - n11 * n24 * n33 - n13 * n21 * n34 + n11 * n23 * n34) * idet;
            ret[3][2] = (n14 * n22 * n31 - n12 * n24 * n31 - n14 * n21 * n32 + n11 * n24 * n32 + n12 * n21 * n34 - n11 * n22 * n34) * idet;
            ret[3][3] = (n12 * n23 * n31 - n13 * n22 * n31 + n13 * n21 * n32 - n11 * n23 * n32 - n12 * n21 * n33 + n11 * n22 * n33) * idet;

            return ret;
        }

        __device__ void identity() {
            memset(m, 0, sizeof(m));
            for (uint32_t i = 0; i < 4; ++i)
                m[i][i] = 1.0f;
        }

        __device__ void rotateY(float angle) {
            identity();
            m[0][0] = cos(angle); 
            m[2][0] = sin(angle);
            m[0][2] = -sin(angle); 
            m[2][2] = cos(angle);
        }
    };
}

namespace scene
{
    struct Sphere
    {
        float4 cr; // center and radius
        float3 color;
        __device__ float3 center() const { return make_float3(cr.x, cr.y, cr.z); }
    };

    struct Plane
    {
        float4 coeff;
        float3 color;
    };

    struct Box
    {
        float3 min, max;
        float3 color;
    };

    constexpr __constant__ float3 ambient{ 20, 20, 20 };
    constexpr __constant__ float3 specularColor{ 255, 255, 255 };
    constexpr __constant__ float shininess = 5.0;
    constexpr __constant__ float3 lightDirection{ -0.3, -0.70710678, -0.70710678 };
    constexpr __constant__ Sphere spheres[1] = {
        Sphere{ {0, 2, -5, 1}, {136, 8, 8} }
    };
    constexpr __constant__ size_t sphereCount = sizeof(spheres) / sizeof(spheres[0]);
    constexpr __constant__ Box boxes[2] = {
        Box {
            {-0.7,-0.7,-0.7}, {0.7, 0.7, 0.7},
            {120, 80, 200}
        },
        Box {
            {-0.7,-0.7,-0.7}, {0.7, 0.7, 0.7},
            {220, 180, 200}
        }
    };
    constexpr __constant__ size_t boxCount = sizeof(boxes) / sizeof(boxes[0]);
    __constant__ Plane planes[1] = {
        Plane { {0, 1, 0, 7}, {50, 50, 70} }
    };
    constexpr __constant__ size_t planeCount = sizeof(planes) / sizeof(planes[0]);
}

namespace math
{
    __device__ float radians(float degrees)
    {
        return degrees * M_PI / 180.0f;
    }

    inline __device__ float3 operator+(float3 a, float3 b)
    {
        return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
    }

    inline __device__ float3 operator-(float3 a)
    {
        return make_float3(-a.x, -a.y, -a.z);
    }

    inline __device__ float3 operator-(float3 a, float3 b)
    {
        return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
    }

    inline __device__ float3 operator*(float3 a, float b)
    {
        return make_float3(a.x * b, a.y * b, a.z * b);
    }

    inline __device__ float3 operator*(float3 a, float3 b)
    {
        return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
    }

    inline __device__ float4 operator*(float4 a, float4 b)
    {
        return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
    }

    inline __device__ float dot(float3 a, float3 b)
    {
        return a.x * b.x + a.y * b.y + a.z * b.z;
    }

    inline __device__ float3 normalize(float3 v)
    {
        return v * rsqrtf(dot(v, v));
    }

    inline __device__ float sign(float x)
    {
        return x > 0 ? 1 : (x < 0 ? -1 : 0);
    }

    inline __device__ float3 sign(float3 x)
    {
        return make_float3(sign(x.x), sign(x.y), sign(x.z));
    }

    inline __device__ float3 abs(float3 v)
    {
        return make_float3(fabsf(v.x), fabsf(v.y), fabsf(v.z));
    }

    inline __device__ float3 pow(float3 v, float k)
    {
        return make_float3(powf(v.x, k), powf(v.y, k), powf(v.z, k));
    }

    inline __device__ float3 step(float3 a, float3 b)
    {
        return make_float3(a.x >= b.x ? 1.0f : 0.0f, a.y >= b.y ? 1.0f : 0.0f, a.z >= b.z ? 1.0f : 0.0f);
    }

    inline __device__ float3 reflect(float3 in, float3 n)
    {
        return in - n * 2.f * dot(in, n);
    }

    __device__ bool intersect(float3 origin, float3 direction, const scene::Sphere& sp, float& t)
    {
        const float3 L = sp.center() - origin;
        const float tca = dot(L, direction);

        if (tca < 0)
            return false;

        const float s2 = (dot(L, L)) - (tca * tca);
        const float s = sqrt(s2);

        if (s > sp.cr.w)
            return false;

        t = tca - sqrt((sp.cr.w * sp.cr.w) - s2);
        return true;
    }

    __device__ float4 intersect(float3 origin, float3 direction, const scene::Box& b)
    {
        const float3 p = origin;

        if ((origin.x < b.max.x && origin.x > b.min.x) &&
            (origin.y < b.max.y && origin.y > b.min.y) &&
            (origin.z < b.max.z && origin.z > b.min.z))
        {
            const float3 center = (b.max + b.min) * 0.5f;
            const float3 n = origin - center;
            const float3 dim = (b.min - b.max) * 0.5f;
            const float bias = 1.01f;
            float3 normal{float(int(n.x / fabsf(dim.x) * bias)), 
            float(int(n.y / fabsf(dim.y) * bias)), 
            float(int(n.z / fabsf(dim.z) * bias))};
            normal = normalize(normal);
            return {0.01f, normal.x, normal.y, normal.z};
        }
        return {-1.0f};
    }

    __device__ float intersect(float3 origin, float3 direction, const scene::Plane& b)
    {
        return -(dot(origin, make_float3(b.coeff.x, b.coeff.y, b.coeff.z)) + b.coeff.w) / dot(direction, make_float3(b.coeff.x, b.coeff.y, b.coeff.z));
    }
}

__device__ void setColor(uint8_t* displayGpu, uint16_t x, uint16_t y, uint8_t r, uint8_t g, uint8_t b)
{
    uint8_t* base = displayGpu + 3 * (y * FB_WIDTH + x);
    base[0] = r;
    base[1] = g;
    base[2] = b;
}

__global__ void raytraceKernel(uint8_t* displayGpu, float globalTime, float C)
{
    using namespace scene;
    using namespace math;

    const auto phongBRDF = [](float3 lightDir, float3 viewDir, float3 normal, float3 phongDiffuseCol, float3 phongSpecularCol, float phongShininess) {
        float3 color = phongDiffuseCol;
        float3 reflectDir = reflect(-lightDir, normal);
        float specDot = fmaxf(dot(reflectDir, viewDir), 0.0);
        color = color + phongSpecularCol * pow(specDot, phongShininess);
        return color;
    };

    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;
    const float pixNormX = (x + 0.5f) / FB_WIDTH;
    const float pixNormY = (y + 0.5f) / FB_HEIGHT;
    const float imageAspectRatio = FB_WIDTH / FB_HEIGHT;
    const float pixRemapX = (2 * pixNormX - 1) * imageAspectRatio;
    const float pixRemapY = 1 - 2 * pixNormY;
    const float pixCameraX = pixRemapX * tan(radians(HORIZONTAL_FOV) / 2);
    const float pixCameraY = pixRemapY * tan(radians(HORIZONTAL_FOV) / 2);

    const float3 camera{ pixCameraX, pixCameraY, -1 };
    const float3 origin{0,0,0};
    float3 dir = normalize(camera - origin);

    const uint32_t iterations = 1280;
    const float maxDistance = 7.0f;
    const float distancePerIteration = maxDistance / iterations;
    const float timePerIteration = distancePerIteration / C;
    for (uint32_t k = 512; k < iterations; ++k)
    {
        const auto localTime = k * timePerIteration;
        const auto time = globalTime - localTime;
        float3 o = origin + dir * distancePerIteration * k;

        for (size_t i = 0; i < sphereCount; ++i)
        {
            const auto scr = spheres[i].cr;

            const Sphere s{ {scr.x + sinf(time), scr.y, scr.z, scr.w}, spheres[i].color };
            float t;
            if (intersect(o, dir, s, t))
            {
                if (t <= distancePerIteration)
                {
                    const float3 hit = origin + dir * t + dir * (k - 1) * distancePerIteration;
                    const float3 normal = normalize(hit - s.center());
                    float3 color = s.color * fmaxf(0.0f, dot(normal, -lightDirection));
                    const float3 R = reflect(-lightDirection, normal);
                    const float specAngle = fmaxf(dot(R, dir), 0.0);
                    const float shininess = 5.0f;
                    const float specular = pow(specAngle, shininess);
                    color = color + float3{ 255, 255, 255 } * specular * 0.4;
                    setColor(displayGpu, x, y, color.x, color.y, color.z);
                    return;
                }
            }
        }

        for (size_t i = 0; i < boxCount; ++i)
        {
            Box b = boxes[i];
            mat4 identity;

            if (i == 0)
            {
                identity.m[3][0] = -2 + sinf(time / 2);
                identity.m[3][1] = -2;
                identity.m[3][2] = -5;
            }
            else
            {
                identity.rotateY(time / 3);
                identity.m[3][0] = 2;
                identity.m[3][1] = -2;
                identity.m[3][2] = -5;
            }
            const mat4 inv = identity.inverse();
            const float4 o4 = make_float4(o.x, o.y, o.z, 1);
            const float4 d4 = make_float4(dir.x, dir.y, dir.z, 0);
            const float4 invO4 = inv * o4;
            const float4 invDir4 = inv * d4;
            const float3 c = (b.min + b.max) * 0.5f;
            const auto oB = float3{ invO4.x, invO4.y, invO4.z };
            const auto dirB = normalize(float3{ invDir4.x, invDir4.y, invDir4.z });

            const float4 maybeHit = intersect(oB, dirB, b);
            if (maybeHit.x > 0)
            {
                float3 baseColor = boxes[i].color;
                const float3 c = (b.min + b.max) * 0.5f;
                const float3 bo{ (oB.x - c.x) * 10.0f, (oB.y - c.y) * 10.0f, (oB.z - c.z) * 10.0f };
                
                if ((int32_t(bo.x) + int32_t(bo.y) + int32_t(bo.z)) % 2 == 0)
                    baseColor = baseColor * 0.1;
                const float3 normal{ maybeHit.y, maybeHit.z, maybeHit.w };
                const float3 viewDir = normalize(origin - o);
                const float3 lightDir = normalize(-lightDirection);
                const float irradiance = fmaxf(dot(lightDir, normal), 0.0);
                float3 radiance = baseColor;
                if (irradiance > 0.0) 
                {
                    const float3 brdf = phongBRDF(lightDir, viewDir, normal, radiance, specularColor, shininess);
                    radiance = radiance + brdf * irradiance;
                }
                const float3 color = pow(radiance, 1.0 / 2.2) * 20;

                setColor(displayGpu, x, y, color.x, color.y, color.z);
                return;
            }
        }
    }


    for (size_t i = 0; i < planeCount; ++i)
    {
        const auto t = intersect(origin, dir, planes[i]);
        if (t > 0.0f)
        {
            const float3 hit = origin + dir * t;
            float3 baseColor = planes[i].color;
            if ((int32_t(hit.x) + int32_t(hit.z)) % 2 == 0)
                baseColor = baseColor * 0.1;
            const float3 normal{ planes[i].coeff.x, planes[i].coeff.y, planes[i].coeff.z };
            const float3 viewDir = normalize(origin - hit);
            const float3 lightDir = normalize(-lightDirection);
            const float irradiance = fmaxf(dot(lightDir, normal), 0.0);
            float3 radiance = ambient;
            if (irradiance > 0.0) 
            {
                const float3 brdf = phongBRDF(lightDir, viewDir, normal, baseColor, specularColor, shininess);
                radiance = radiance + brdf * irradiance;
            }

            float3 color = pow(radiance, 1.0 / 2.2) * 10;
            const float timeToPlaneHit = t / C;
            for (uint32_t k = 0; k < iterations; ++k)
            {
                const auto localTime = k * timePerIteration;
                const auto time = globalTime - localTime - timeToPlaneHit;
                float3 o = hit - lightDirection * distancePerIteration * k * 3.0f;

                bool hitFound{};
                for (size_t i = 0; i < sphereCount; ++i)
                {
                    const auto scr = spheres[i].cr;
                    const Sphere s{ {scr.x + sinf(time), scr.y, scr.z, scr.w}, {} };

                    float t;
                    if (intersect(o, -lightDirection, s, t))
                    {
                        if (t <= distancePerIteration)
                        {
                            color = color * 0.5;
                            hitFound = true;
                            break;
                        }
                    }
                }
                if (hitFound)
                    break;

                for (size_t i = 0; i < boxCount; ++i)
                {
                    Box b = boxes[i];
                    auto oB = hit - lightDirection * distancePerIteration * k * 3.0f;;
                    auto dirB = -lightDirection;
                    mat4 identity;

                    if (i == 0)
                    {
                        identity.m[3][0] = -2 + sinf(time / 2);
                        identity.m[3][1] = -2;
                        identity.m[3][2] = -5;
                    }
                    else
                    {
                        identity.rotateY(time / 3);
                        identity.m[3][0] = 2;
                        identity.m[3][1] = -2;
                        identity.m[3][2] = -5;
                    }
                    const mat4 inv = identity.inverse();
                    const float4 o4 = make_float4(oB.x, oB.y, oB.z, 1);
                    const float4 d4 = make_float4(dirB.x, dirB.y, dirB.z, 0);
                    const float4 invO4 = inv * o4;
                    const float4 invDir4 = inv * d4;
                    const float3 c = (b.min + b.max) * 0.5f;
                    oB = float3{ invO4.x, invO4.y, invO4.z };
                    dirB = normalize(float3{ invDir4.x, invDir4.y, invDir4.z });

                    const float4 maybeHit = intersect(oB, dirB, b);
                    if (maybeHit.x > 0)
                    {
                        color = color * 0.5;
                        hitFound = true;
                        break;
                    }
                }

                if (hitFound)
                    break;
            }

            setColor(displayGpu, x, y, color.x, color.y, color.z);
            return;
        }
    }

    setColor(displayGpu, x, y, 0, 0, 0);
}

void setup_logging()
{
    spdlog::set_pattern("[%c %z] [%^%L%$] %v");
    spdlog::set_level(spdlog::level::debug);
}

int main(int, char**)
{
    setup_logging();
    bool quit = false;

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
        spdlog::error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

    SDL_Event event;
    SDL_Init(SDL_INIT_VIDEO);
    SDL_Window* window = SDL_CreateWindow("rt-sol", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, WINDOW_WIDTH, WINDOW_HEIGHT, 0);
    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, 0);
    SDL_RendererInfo info{};
    SDL_GetRendererInfo(renderer, &info);
    SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, FB_WIDTH, FB_HEIGHT);

    std::vector<uint8_t> display;
    uint8_t* displayGpu{};
    display.resize(FB_WIDTH * FB_HEIGHT * 3);

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&displayGpu, FB_HEIGHT * FB_WIDTH * 3);
    if (cudaStatus != hipSuccess)
        spdlog::error("hipMalloc failed!");

    float t{};
    float C{ 0.5f };
    while (!quit)
    {
        const auto t1 = std::chrono::high_resolution_clock::now();
        raytraceKernel << <BLOCKS, THREADS >> > (displayGpu, t, C);
        t += 0.032;

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
            spdlog::error("emulateKernel launch failed: {}", hipGetErrorString(cudaStatus));

        const auto t2 = std::chrono::high_resolution_clock::now();
        //spdlog::info("CUDA: {}ms", std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count());

        hipMemcpy(display.data(), displayGpu, FB_HEIGHT * FB_WIDTH * 3, hipMemcpyDeviceToHost);

        SDL_RenderCopy(renderer, texture, NULL, NULL);
        SDL_RenderPresent(renderer);
        SDL_UpdateTexture(texture, nullptr, display.data(), FB_WIDTH * 3);

        while (SDL_PollEvent(&event))
        {
            SDL_PumpEvents();
            quit = event.type == SDL_QUIT;
        }

        int len{};
        const uint8_t* keys = SDL_GetKeyboardState(&len);
        if (keys[SDL_SCANCODE_UP])
        {
            C += 0.05f;
            spdlog::info("C = {}", C);
        }
        else if (keys[SDL_SCANCODE_DOWN])
        {
            C -= 0.05f;
            spdlog::info("C = {}", C);
        }

        // 60hz
        using namespace std::chrono_literals;
        //std::this_thread::sleep_for(16ms);
    }

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    hipFree(displayGpu);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}